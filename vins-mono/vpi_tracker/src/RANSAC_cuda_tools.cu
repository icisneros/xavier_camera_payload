#include "hip/hip_runtime.h"
#include "RANSAC_cuda_tools.h"
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
namespace RANSAC_cuda_tools{


// __constant__ float parameters[10*INITIAL_ITER];

// compute fundamental mat, use shared memory
// find null space for fundamental mat, the dimension should be 1
void test(){
    std::cerr<<"hello world"<<std::endl;
}
__inline__  __device__ void find_null(float* matrix, int row, int col,float *null_space, bool &success)
{
    int pivate_idx = 0;
    success = true;
                    // printf(
                    //     "[%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                    //     "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                    //     "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f ]\n",
                    //     matrix[0*col],matrix[0*col+1],matrix[0*col+2],matrix[0*col+3],matrix[0*col+4],matrix[0*col+5],matrix[0*col+6],matrix[0*col+7],matrix[0*col+8],
                    //     matrix[1*col],matrix[1*col+1],matrix[1*col+2],matrix[1*col+3],matrix[1*col+4],matrix[1*col+5],matrix[1*col+6],matrix[1*col+7],matrix[1*col+8],
                    //     matrix[2*col],matrix[2*col+1],matrix[2*col+2],matrix[2*col+3],matrix[2*col+4],matrix[2*col+5],matrix[2*col+6],matrix[2*col+7],matrix[2*col+8]);
    // reduced to row echelon form
    for (int i=0;i<row-1;i++){
        // printf("pivate row %d\n",i);
        float pivate = matrix[col*i+pivate_idx];
        if (fabs(pivate) >1e-9){
            for (int j=i+1;j<row;j++){
                float m = matrix[col*j+pivate_idx]/pivate;
                matrix[col*j+pivate_idx] = 0;
                for (int k=pivate_idx+1;k<col;k++){
                    matrix[col*j+k] = matrix[col*j+k]-m*matrix[col*i+k];
                }
            }
        }
        else {
            success = false;
            break;
        }
        pivate_idx = pivate_idx + 1;
    }
    // __syncthreads();
    //find null space
    // visualize matrix
        // printf("[%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
        //                 "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
        //                 "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
        //                 "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
        //                 "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
        //                 "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
        //                 "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
        //                 "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f ]\n",
        //                 matrix[0*col],matrix[0*col+1],matrix[0*col+2],matrix[0*col+3],matrix[0*col+4],matrix[0*col+5],matrix[0*col+6],matrix[0*col+7],matrix[0*col+8],
        //                 matrix[1*col],matrix[1*col+1],matrix[1*col+2],matrix[1*col+3],matrix[1*col+4],matrix[1*col+5],matrix[1*col+6],matrix[1*col+7],matrix[1*col+8],
        //                 matrix[2*col],matrix[2*col+1],matrix[2*col+2],matrix[2*col+3],matrix[2*col+4],matrix[2*col+5],matrix[2*col+6],matrix[2*col+7],matrix[2*col+8],
        //                 matrix[3*col],matrix[3*col+1],matrix[3*col+2],matrix[3*col+3],matrix[3*col+4],matrix[3*col+5],matrix[3*col+6],matrix[3*col+7],matrix[3*col+8],
        //                 matrix[4*col],matrix[4*col+1],matrix[4*col+2],matrix[4*col+3],matrix[4*col+4],matrix[4*col+5],matrix[4*col+6],matrix[4*col+7],matrix[4*col+8],
        //                 matrix[5*col],matrix[5*col+1],matrix[5*col+2],matrix[5*col+3],matrix[5*col+4],matrix[5*col+5],matrix[5*col+6],matrix[5*col+7],matrix[5*col+8],
        //                 matrix[6*col],matrix[6*col+1],matrix[6*col+2],matrix[6*col+3],matrix[6*col+4],matrix[6*col+5],matrix[6*col+6],matrix[6*col+7],matrix[6*col+8],
        //                 matrix[7*col],matrix[7*col+1],matrix[7*col+2],matrix[7*col+3],matrix[7*col+4],matrix[7*col+5],matrix[7*col+6],matrix[7*col+7],matrix[7*col+8]);
                // printf(
                //         "[%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                //         "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                //         "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f ]\n",
                //         matrix[0*col],matrix[0*col+1],matrix[0*col+2],matrix[0*col+3],matrix[0*col+4],matrix[0*col+5],matrix[0*col+6],matrix[0*col+7],matrix[0*col+8],
                //         matrix[1*col],matrix[1*col+1],matrix[1*col+2],matrix[1*col+3],matrix[1*col+4],matrix[1*col+5],matrix[1*col+6],matrix[1*col+7],matrix[1*col+8],
                //         matrix[2*col],matrix[2*col+1],matrix[2*col+2],matrix[2*col+3],matrix[2*col+4],matrix[2*col+5],matrix[2*col+6],matrix[2*col+7],matrix[2*col+8]);

                        // printf("                                                                 \n"
                        // "[%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                        // "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                        // "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f ]\n",
                        // matrix[5*col],matrix[5*col+1],matrix[5*col+2],matrix[5*col+3],matrix[5*col+4],matrix[5*col+5],matrix[5*col+6],matrix[5*col+7],matrix[5*col+8],
                        // matrix[6*col],matrix[6*col+1],matrix[6*col+2],matrix[6*col+3],matrix[6*col+4],matrix[6*col+5],matrix[6*col+6],matrix[6*col+7],matrix[6*col+8],
                        // matrix[7*col],matrix[7*col+1],matrix[7*col+2],matrix[7*col+3],matrix[7*col+4],matrix[7*col+5],matrix[7*col+6],matrix[7*col+7],matrix[7*col+8]);
    if (success){
        // int back_col = col-1;
        // null_space[back_col--] = 1;
        // while (back_col>0){
        //     for (int i=back_col+1;i<col;i++){
        //         null_space[back_col] = null_space[back_col]-matrix[back_col*row+i]*null_space[i];
        //     }
        //     null_space[back_col--] /= matrix[back_col*row+back_col];
        // }
        null_space[8] = 1;
        null_space[7] = -matrix[71]*null_space[8]/matrix[70];
        null_space[6] = -(matrix[62]*null_space[8]+matrix[61]*null_space[7])/matrix[60];
        null_space[5] = -(matrix[53]*null_space[8]+matrix[52]*null_space[7]+matrix[51]*null_space[6])/matrix[50];
        null_space[4] = -(matrix[44]*null_space[8]+matrix[43]*null_space[7]+matrix[42]*null_space[6]+matrix[41]*null_space[5])/matrix[40];
        null_space[3] = -(matrix[35]*null_space[8]+matrix[34]*null_space[7]+matrix[33]*null_space[6]+matrix[32]*null_space[5]+matrix[31]*null_space[4])/matrix[30];
        null_space[2] = -(matrix[26]*null_space[8]+matrix[25]*null_space[7]+matrix[24]*null_space[6]+matrix[23]*null_space[5]+matrix[22]*null_space[4]+matrix[21]*null_space[3])/matrix[20];
        null_space[1] = -(matrix[17]*null_space[8]+matrix[16]*null_space[7]+matrix[15]*null_space[6]+matrix[14]*null_space[5]+matrix[13]*null_space[4]+matrix[12]*null_space[3]+matrix[11]*null_space[2])/matrix[10];
        null_space[0] = -(matrix[8] * null_space[8]+ matrix[7] *null_space[7]+matrix[6]*null_space[6]+matrix[5]*null_space[5]+matrix[4]*null_space[4]+matrix[3]*null_space[3]+matrix[2]*null_space[2]+matrix[1]*null_space[1])/matrix[0];
        // int back_col = col-1;
    }
    // __syncthreads();
}

/*cur_d: cur pts
    forw_d: forw_pts
    out_null_space: 10*INIT_ITER, first element indicate success state

    one thread uses one group of eight pairs of points
    one thread calculates one model 
    in total 512 threads, i.e. threads' global indices (1-D) indicate which model to compute: iy *  
    each block contains pts.size() threads since we want to use shared memory to accelerate data fetching
    one thread copy one pairs of points to shared memory (dynamic)

*/
__global__ void computeFundamentalMat(float* cur_d,float* forw_d,int max_pts,hiprandStatePhilox4_32_10_t *state,float *out_null_space)
{
    // first max_pts * 2: cur_d
    // second max_pts *2: forw_d
    extern __shared__ float points[];
    // extern __shared__ float s_forw[];
    //copy data to shared memory to reduce latency
    // coordinate inside the memory
    int pts_idx = threadIdx.x + blockDim.x*threadIdx.y;
    points[pts_idx] = cur_d[pts_idx];
    points[pts_idx+max_pts] = cur_d[pts_idx+max_pts];
    points[pts_idx+max_pts*2] = forw_d[pts_idx];
    points[pts_idx+max_pts*3] = forw_d[pts_idx+max_pts];
    __syncthreads();

    // int ix = threadIdx.x+blockIdx.x*blockDim.x;
    // int iy = threadIdx.y+blockIdx.y*blockDim.y;
    // global coordinate, indicate model number

    int idx = threadIdx.x+blockIdx.x*blockDim.x + (threadIdx.y+blockIdx.y*blockDim.y)*gridDim.x*blockDim.x;
    int seed = idx;
    float4 randomidx1;
    float4 randomidx2;
    hiprand_init(seed, idx, 0, &state[idx]);
    randomidx1 = hiprand_uniform4(&state[idx]);
    randomidx2 = hiprand_uniform4(&state[idx]);
    int indices[8] = {(int)(randomidx1.x*max_pts),(int)(randomidx1.y*max_pts),(int)(randomidx1.z*max_pts)
    ,(int)(randomidx1.w*max_pts),(int)(randomidx2.x*max_pts),(int)(randomidx2.y*max_pts),(int)(randomidx2.z*max_pts),(int)(randomidx2.w*max_pts)};
    // printf("model %d random idx %d %d %d %d %d %d %d %d\n",idx,indices[0],indices[1],indices[2],indices[3],indices[4],indices[5],indices[6],indices[7]);
    // float* cur_row1 = (float*)((char*)cur_d);
    // float* cur_row2 = (float*)((char*)cur_d + cur_pitch);
    // float* forw_row1 = (float*)((char*)forw_d);
    // float* forw_row2 = (float*)((char*)forw_d + forw_pitch);
    // printf("pts_idx:%d cur_pt(%.3f, %.3f) forw_pt(%.3f, %.3f)\n",pts_idx,points[pts_idx], points[pts_idx+max_pts],points[pts_idx+max_pts*2], points[pts_idx+max_pts*3]);
    if (idx<INITIAL_ITER){
        float matrix[72];
        // we do not use a loop here to reduce latency
        //first row
        int p_idx = indices[0];
        float u1 = points[p_idx];
        float v1 = points[p_idx+max_pts];
        float u2 = points[p_idx+max_pts*2];
        float v2 = points[p_idx+max_pts*3];
        matrix[0] = u1*u2;
        matrix[1] = u1*v2;
        matrix[2] = u1;                   
        matrix[3] = v1*u2;
        matrix[4] = v1*v2;
        matrix[5] = v1;
        matrix[6] = u2;
        matrix[7] = v2;
        matrix[8] = 1;

        //second row
        p_idx = indices[1];
        u1 = points[p_idx];
        v1 = points[p_idx+max_pts];
        u2 = points[p_idx+max_pts*2];
        v2 = points[p_idx+max_pts*3];
        matrix[9] = u1*u2;
        matrix[10] = u1*v2;
        matrix[11] = u1;                   
        matrix[12] = v1*u2;
        matrix[13] = v1*v2;
        matrix[14] = v1;
        matrix[15] = u2;
        matrix[16] = v2;
        matrix[17] = 1;
        //third row
        p_idx = indices[2];
        u1 = points[p_idx];
        v1 = points[p_idx+max_pts];
        u2 = points[p_idx+max_pts*2];
        v2 = points[p_idx+max_pts*3];
        matrix[18] = u1*u2;
        matrix[19] = u1*v2;
        matrix[20] = u1;                   
        matrix[21] = v1*u2;
        matrix[22] = v1*v2;
        matrix[23] = v1;
        matrix[24] = u2;
        matrix[25] = v2;
        matrix[26] = 1;
        //forth row
        p_idx = indices[3];
        u1 = points[p_idx];
        v1 = points[p_idx+max_pts];
        u2 = points[p_idx+max_pts*2];
        v2 = points[p_idx+max_pts*3];
        matrix[27] = u1*u2;
        matrix[28] = u1*v2;
        matrix[29] = u1;                   
        matrix[30] = v1*u2;
        matrix[31] = v1*v2;
        matrix[32] = v1;
        matrix[33] = u2;
        matrix[34] = v2;
        matrix[35] = 1;
        //fifth row
        p_idx = indices[4];
        u1 = points[p_idx];
        v1 = points[p_idx+max_pts];
        u2 = points[p_idx+max_pts*2];
        v2 = points[p_idx+max_pts*3];
        matrix[36] = u1*u2;
        matrix[37] = u1*v2;
        matrix[38] = u1;                   
        matrix[39] = v1*u2;
        matrix[40] = v1*v2;
        matrix[41] = v1;
        matrix[42] = u2;
        matrix[43] = v2;
        matrix[44] = 1;
        //sixth row
        p_idx = indices[5];
        u1 = points[p_idx];
        v1 = points[p_idx+max_pts];
        u2 = points[p_idx+max_pts*2];
        v2 = points[p_idx+max_pts*3];
        matrix[45] = u1*u2;
        matrix[46] = u1*v2;
        matrix[47] = u1;                   
        matrix[48] = v1*u2;
        matrix[49] = v1*v2;
        matrix[50] = v1;
        matrix[51] = u2;
        matrix[52] = v2;
        matrix[53] = 1;
        //seventh row
        p_idx = indices[6];
        u1 = points[p_idx];
        v1 = points[p_idx+max_pts];
        u2 = points[p_idx+max_pts*2];
        v2 = points[p_idx+max_pts*3];
        matrix[54] = u1*u2;
        matrix[55] = u1*v2;
        matrix[56] = u1;                   
        matrix[57] = v1*u2;
        matrix[58] = v1*v2;
        matrix[59] = v1;
        matrix[60] = u2;
        matrix[61] = v2;
        matrix[62] = 1;
        //eighth row
        p_idx = indices[7];
        u1 = points[p_idx];
        v1 = points[p_idx+max_pts];
        u2 = points[p_idx+max_pts*2];
        v2 = points[p_idx+max_pts*3];
        matrix[63] = u1*u2;
        matrix[64] = u1*v2;
        matrix[65] = u1;                   
        matrix[66] = v1*u2;
        matrix[67] = v1*v2;
        matrix[68] = v1;
        matrix[69] = u2;
        matrix[70] = v2;
        matrix[71] = 1;

        // __syncthreads();

        // printf cannot support too much elements, print at most 3 rows
                    // printf("#######################################\n"
                    //     "pts_idx:%d global idx:%d\n"
                    //     "[%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                    //     "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                    //     "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                    //     "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                    //     "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                    //     "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                    //     "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                    //     "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f ]\n"
                    //     "###############################################\n",
                    //     pts_idx,idx,
                    //     matrix[0*9],matrix[0*9+1],matrix[0*9+2],matrix[0*9+3],matrix[0*9+4],matrix[0*9+5],matrix[0*9+6],matrix[0*9+7],matrix[0*9+8],
                    //     matrix[1*9],matrix[1*9+1],matrix[1*9+2],matrix[1*9+3],matrix[1*9+4],matrix[1*9+5],matrix[1*9+6],matrix[1*9+7],matrix[1*9+8],
                    //     matrix[2*9],matrix[2*9+1],matrix[2*9+2],matrix[2*9+3],matrix[2*9+4],matrix[2*9+5],matrix[2*9+6],matrix[2*9+7],matrix[2*9+8],
                    //     matrix[3*9],matrix[3*9+1],matrix[3*9+2],matrix[3*9+3],matrix[3*9+4],matrix[3*9+5],matrix[3*9+6],matrix[3*9+7],matrix[3*9+8],
                    //     matrix[4*9],matrix[4*9+1],matrix[4*9+2],matrix[4*9+3],matrix[4*9+4],matrix[4*9+5],matrix[4*9+6],matrix[4*9+7],matrix[4*9+8],
                    //     matrix[5*9],matrix[5*9+1],matrix[5*9+2],matrix[5*9+3],matrix[5*9+4],matrix[5*9+5],matrix[5*9+6],matrix[5*9+7],matrix[5*9+8],
                    //     matrix[6*9],matrix[6*9+1],matrix[6*9+2],matrix[6*9+3],matrix[6*9+4],matrix[6*9+5],matrix[6*9+6],matrix[6*9+7],matrix[6*9+8],
                    //     matrix[7*9],matrix[7*9+1],matrix[7*9+2],matrix[7*9+3],matrix[7*9+4],matrix[7*9+5],matrix[7*9+6],matrix[7*9+7],matrix[7*9+8]);
        float null_space[9] = {0,0,0,0,0,0,0,0,0};
        bool success = true;

        find_null(matrix,8,9,null_space,success);
        //                                         printf("\n"
        // "global idx%d \n"
        // "(u1,v1): (%.2f,%.2f) (u2,v2): (%.2f,%.2f)\n",
        // idx, u1,v1,u2,v2);
        // int col = 9;
                        // printf("                                                                 \n"
                        // "matrix for model %d:\n"
                        // "[%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                        // "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f \n"
                        // "%0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f %0.2f ]\n",
                        // idx,
                        // matrix[5*col],matrix[5*col+1],matrix[5*col+2],matrix[5*col+3],matrix[5*col+4],matrix[5*col+5],matrix[5*col+6],matrix[5*col+7],matrix[5*col+8],
                        // matrix[6*col],matrix[6*col+1],matrix[6*col+2],matrix[6*col+3],matrix[6*col+4],matrix[6*col+5],matrix[6*col+6],matrix[6*col+7],matrix[6*col+8],
                        // matrix[7*col],matrix[7*col+1],matrix[7*col+2],matrix[7*col+3],matrix[7*col+4],matrix[7*col+5],matrix[7*col+6],matrix[7*col+7],matrix[7*col+8]);
        out_null_space[idx*10] = (float)success;
        // for (int i=1;i<10;i++){
        out_null_space[idx*10+1] = null_space[0];
        out_null_space[idx*10+2] = null_space[1];
        out_null_space[idx*10+3] = null_space[2];
        out_null_space[idx*10+4] = null_space[3];
        out_null_space[idx*10+5] = null_space[4];
        out_null_space[idx*10+6] = null_space[5];
        out_null_space[idx*10+7] = null_space[6];
        out_null_space[idx*10+8] = null_space[7];
        out_null_space[idx*10+9] = null_space[8];
        // }

    }
    // __syncthreads();
    //remember to copy the null space parameters to constant memory in host
}
// test the model on each thread, use shared memory
// each block runs one model (512 blocks), each thread test one data (pts thread each)
// points value stored in shared memory
//512*pts_size
__global__ void testModel(float* cur_d,float* forw_d, int max_pts,float threashold,int* inlier_count,float* parameters,uchar* statuses)
{
    // first max_pts * 2: cur_d
    // second max_pts *2: forw_d
    extern __shared__ float points[];
    // extern __shared__ float s_forw[];
    //copy data to shared memory to reduce latency
    // coordinate inside the memory
    int pts_idx = threadIdx.x + blockDim.x*threadIdx.y;
    points[pts_idx] = cur_d[pts_idx];
    points[pts_idx+max_pts] = cur_d[pts_idx+max_pts];
    points[pts_idx+max_pts*2] = forw_d[pts_idx];
    points[pts_idx+max_pts*3] = forw_d[pts_idx+max_pts];
    points[max_pts*4] = 0;
    __syncthreads();
    int model_idx = blockIdx.x + blockDim.x*blockIdx.y;
    if (parameters[10*model_idx] != 0){
            //verify the point
            float u1 = points[pts_idx];
            float v1 = points[pts_idx+max_pts];
            float u2 = points[pts_idx+max_pts*2];
            float v2 = points[pts_idx+max_pts*3];
            float my_vec[9] = {u1*u2,u1*v2,u1,v1*u2,v1*v2,v1,u2,v2,1};
            // float* cur_row1 = (float*)((char*)cur_d);
            // float* cur_row2 = (float*)((char*)cur_d + cur_pitch);
            // float* forw_row1 = (float*)((char*)forw_d);
            // float* forw_row2 = (float*)((char*)forw_d + forw_pitch);
            float result = 0;
            result += my_vec[0]*parameters[10*model_idx + 1];
            result += my_vec[1]*parameters[10*model_idx + 2];
            result += my_vec[2]*parameters[10*model_idx + 3];
            result += my_vec[3]*parameters[10*model_idx + 4];
            result += my_vec[4]*parameters[10*model_idx + 5];
            result += my_vec[5]*parameters[10*model_idx + 6];
            result += my_vec[6]*parameters[10*model_idx + 7];
            result += my_vec[7]*parameters[10*model_idx + 8];
            result += my_vec[8]*parameters[10*model_idx + 9];
            statuses[model_idx*max_pts+pts_idx] = (uchar)(fabs(result)<threashold);
            atomicAdd(&points[max_pts*4],(int)(fabs(result)<threashold));
            // atomicAdd(&inlier_count[model_idx],1);
            // }
    }
    inlier_count[model_idx] = points[max_pts*4];
    // __syncthreads();
}

__host__ void findFundamentalMat_on_cuda(std::vector<cv::Point2f> &i1, std::vector<cv::Point2f> &i2,double threashold, double confidence,std::vector<uchar> & status){
    //Generate M sets of random indices in CPU eight-point-algorithm
    //M: number of iters
    if (i1.size()<32){
        cv::findFundamentalMat(i1, i2, cv::FM_RANSAC, 1,confidence, status);
        return;
    }
    #if DEBUGGING
    std::chrono::time_point<std::chrono::high_resolution_clock> startt, endt;
    startt = std::chrono::high_resolution_clock::now();
    std::cerr<<"init ransac parameters"<<std::endl;
    #endif

    int s = i1.size();
    int global_max_inlier = 0;
    int max_idx = 0;
    float* bestParameter = (float*)malloc(sizeof(float)*9);
    int repeatance = 0;
    // std::vector<unsigned> indices;
    // for (int i =0;i<s;i++){
    //         indices.push_back(i);
    // }
    // int endIdx = indices.size()-8*((int)(indices.size()/8));
    // Copy the points to GPU
    // std::cerr<<"init pts data on GPU"<<std::endl;
    std::chrono::time_point<std::chrono::high_resolution_clock> cvstartt, cvendt;
    cvstartt = std::chrono::high_resolution_clock::now();
    int width = s, height = 2;
    float*cur_pts_d;
    float* forw_pts_d;
    hipMalloc(&cur_pts_d,sizeof(float)*width*height);
    hipMalloc(&forw_pts_d,sizeof(float)*width*height);
    // size_t pitch1, pitch2;
    // size_t h_pitch = width*sizeof(float);
    // hipMallocPitch(&cur_pts_d,&pitch1,width*sizeof(float),height);
    // hipMallocPitch(&forw_pts_d,&pitch2,width*sizeof(float),height);
    // std::cerr<<"copy pts:"<<s<<std::endl;
    float *forw_pts_h = (float*)malloc(sizeof(float)*width*height);
    float *cur_pts_h = (float*)malloc(sizeof(float)*width*height);
    for (int i =0;i<s;i++){
        cur_pts_h[i] = i1[i].x;
        cur_pts_h[s+i] = i1[i].y;
        forw_pts_h[i] = i2[i].x;
        forw_pts_h[s+i] = i2[i].y;
    }
    hipMemcpy(cur_pts_d,cur_pts_h,sizeof(float)*width*height,hipMemcpyHostToDevice);
    hipMemcpy(forw_pts_d,forw_pts_h,sizeof(float)*width*height,hipMemcpyHostToDevice);
    // CHECK_CUDA(hipMemcpy2D(cur_pts_d,pitch1,cur_pts_h,h_pitch,width*sizeof(float),height,hipMemcpyHostToDevice));
    // CHECK_CUDA(hipMemcpy2D(forw_pts_d,pitch1,forw_pts_h,h_pitch,width*sizeof(float),height,hipMemcpyHostToDevice));
    free(cur_pts_h);
    free(forw_pts_h);
    cvendt = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cvelapsed_seconds = cvendt-cvstartt;
    // std::cout<<"Copying feature points to CUDA costs:"<<std::endl<<std::fixed<<std::setprecision(6)<<cvelapsed_seconds.count() <<" seconds"<<std::endl;
    float* null_space_h = (float*)malloc(sizeof(float)*10*INITIAL_ITER);
    float* null_space_d;
    hipMalloc(&null_space_d,sizeof(float)*10*INITIAL_ITER);
    int* inlier_count_d;
    int* inlier_count_h = (int*)malloc(sizeof(int)*INITIAL_ITER);
    hipMalloc(&inlier_count_d,sizeof(int)*INITIAL_ITER);
    uchar* statuses;
    hipMalloc(&statuses,sizeof(uchar)*s*INITIAL_ITER);
    // std::cerr<<"enter while loop"<<std::endl;
    //main loop to run each parallel operation 512 itteration each
    #ifdef DEBUGGING
        endt = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_seconds = endt-startt;
        std::cout<<"Allocate memory on CUDA costs:"<<std::endl<<std::fixed<<std::setprecision(6)<<elapsed_seconds.count() <<" seconds"<<std::endl;
    #endif
    while (true){

        #ifdef DEBUGGING
        startt = std::chrono::high_resolution_clock::now();
        #endif
        bool flip = false;
        hiprandStatePhilox4_32_10_t  *dev_random;
        hipMalloc((void**)&dev_random,s*((int)(INITIAL_ITER/s)+1)*sizeof(hiprandState));
        // std::chrono::duration<double> elapsed_seconds = endt-startt;
        computeFundamentalMat<<<(int)(INITIAL_ITER/s)+1,s,sizeof(float)*s*2*2>>>(cur_pts_d,forw_pts_d,s,dev_random,null_space_d);
        // hipDeviceSynchronize();
        hipFree(dev_random);
        // std::cerr<<"finish compute"<<std::endl;
        // CHECK_CUDA(hipFree(randomIdx));
        // CHECK_CUDA(hipMemcpy(null_space_h,null_space_d,sizeof(float)*10*INITIAL_ITER,hipMemcpyDeviceToHost));
        hipMemcpy(inlier_count_d,inlier_count_h,sizeof(int)*INITIAL_ITER,hipMemcpyHostToDevice);
        // CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(parameters),null_space_h,sizeof(float)*10*INITIAL_ITER));
        // std::cerr<<"test Model"<<std::endl;
        testModel<<<INITIAL_ITER,s,sizeof(float)*(s*2*2+1)>>>(cur_pts_d,forw_pts_d,s,threashold,inlier_count_d,null_space_d,statuses);
        //  hipDeviceSynchronize();
        //  std::cerr<<"memcpy to host"<<std::endl;
        hipMemcpy(inlier_count_h,inlier_count_d,sizeof(int)*INITIAL_ITER,hipMemcpyDeviceToHost);
        #ifdef DEBUGGING
        endt = std::chrono::high_resolution_clock::now();
        elapsed_seconds = endt-startt;
        std::cout<<"Computing fundamental matrices and verification models costs:"<<std::endl<<std::fixed<<std::setprecision(6)<<elapsed_seconds.count() <<" seconds"<<std::endl;
        #endif
        int max_inlier = global_max_inlier;

        // std::cout<<std::endl;
        for (int i=0;i<INITIAL_ITER;i++){
            // std::cout<<inlier_count_h[i]<<',';
            if (inlier_count_h[i]>max_inlier){
                max_idx = i;
                max_inlier = inlier_count_h[i];
                flip = true;
            }
        }
        if (flip){
            global_max_inlier = max_inlier;
            // CHECK_CUDA(hipMemcpy(bestParameter,null_space_d+max_idx*10+1,sizeof(float)*9,hipMemcpyDeviceToHost));
            // for (int k=0;k<9;k++){
            //     bestParameter[k] = null_space_h[10*max_idx+1+k];
            // }
            // CHECK_CUDA(hipMemcpyFromSymbol(bestParameter,HIP_SYMBOL(parameters),sizeof(float)*9,(max_idx*10+1)*sizeof(float)));
                // std::cout<<std::endl;
            // for (int i =0;i<9;i++){
            //     std::cout<<bestParameter[i]<<',';
            // }
        }
        // std::cout<<std::endl<<"best idx"<<max_idx<<std::endl;
        // std::cout<<"max inlier:"<<max_inlier<<"/"<<s<<std::endl;
        // compute iteration time 
        confidence = MAX(confidence, 0.);
        confidence = MIN(confidence, 1.);
        float numerator = std::log(1-confidence);
        // std::cerr<<"numerator"<< numerator<<std::endl;
        // std::cerr<<((float)max_inlier/s)<<std::endl;
        float denominator = std::log(1-std::pow((float)max_inlier/s,8));
        // std::cerr<<"denominator"<< denominator<<std::endl;
        // std::cerr<<"calculated iter"<< cvRound(numerator/denominator)<<std::endl;
        int iter = denominator>=0 || -numerator>= MAX_ITER*(-denominator) ? MAX_ITER : cvRound(numerator/denominator);
        // std::cerr<<"mim iter num"<<iter<<std::endl;
        repeatance++;
        if (repeatance*INITIAL_ITER>iter) break;
    }

    hipFree(cur_pts_d);
    hipFree(forw_pts_d);
    hipFree(null_space_d);
    hipFree(inlier_count_d);
    free(inlier_count_h);
    free(bestParameter);
    free(null_space_h);
    // startt = std::chrono::high_resolution_clock::now();
    uchar *status_h  = (uchar*)malloc(sizeof(uchar)*s);
    hipMemcpy(status_h,statuses+max_idx*s,sizeof(uchar)*s,hipMemcpyDeviceToHost);
        hipFree(statuses);
    // std::cout<<"result"<<std::endl;
    status.resize(s);
    for (int i=0;i<s;i++){
        // std::cout<<"point:"<<i<<":"<<(int)status_h[i]<<std::endl;
        status[i] = status_h[i];
        // float u1 = i1[i].x;
        // float v1 = i1[i].y;
        // float u2 = i2[i].x;
        // float v2 = i2[i].y;
        // float my_vec[9] = {u1*u2,u1*v2,u1,v1*u2,v1*v2,v1,u2,v2,1};
        // float result=0;
        // for (int j=0;j<9;j++){
        //     result += my_vec[j]*bestParameter[j];
        // }
        // // std::cout<<"(pt"<<i<<":"<<fabs(result)<<"),";
        // if (fabs(result) < threashold){
        //     status[i] = 1;
        // }
        // else{
        //     status[i] = 0;
        //  }    
    }
    //  std::cout<<"result"<<std::endl;
    #if DEBUGGING
    // endt = std::chrono::high_resolution_clock::now();
    startt = std::chrono::high_resolution_clock::now();
    #endif
    // hipFree(statuses);
    // hipFree(cur_pts_d);
    // hipFree(forw_pts_d);
    // hipFree(null_space_d);
    // hipFree(inlier_count_d);
    // free(inlier_count_h);
    // free(bestParameter);
    // free(null_space_h);
    free(status_h);
    #if DEBUGGING
        endt = std::chrono::high_resolution_clock::now();
        elapsed_seconds = endt-startt;
        std::cout<<"Free memory costs:"<<std::endl<<std::fixed<<std::setprecision(6)<<elapsed_seconds.count() <<" seconds"<<std::endl;
    #endif
}
}